#include "hip/hip_runtime.h"
/*
	The current approach uses only texture memory. Performance is way better than global memory.
	Future release might contain share memory embedded in it. :D
*/

#include "../API/gpu_filter_api.h"
#include "hip/hip_runtime.h"
#include "assert.h"
#include "stdio.h"
#include "math.h"

#define MAD(a, b, c) ( __mul24((a), (b)) + (c) )
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

inline int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

__constant__ float Kernel[33]; // Using constant memory due to high bandwidth availabilitity

texture<unsigned char, 2, hipReadModeElementType> texSrc; // Declaring texture memory

//////////////////////////////////////////////// Row convolution filter ///////////////////////////////////////////////////
__global__ void convolutionRowsKernel( unsigned char *dst, int imageW, int imageH, int KERNEL_RADIUS, int KERNEL_LENGTH)
{
    const   int ix = MAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = MAD(blockDim.y, blockIdx.y, threadIdx.y);
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

    if(ix >= imageW || iy >= imageH)
        return;

    float sum = 0;

   /* #if(UNROLL_INNER)
        sum = convolutionRow<2 * KERNEL_RADIUS>(x, y);
    #else*/
    #pragma unroll
        for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
        sum += tex2D(texSrc, x + (float)k, y) * Kernel[KERNEL_RADIUS - k];
  //  #endif

    dst[MAD(iy, imageW, ix)] = (unsigned char)sum;
}


void convolutionRowsGPU( unsigned char *dst, hipArray *src, int imageW, int imageH, int KERNEL_RADIUS, int KERNEL_LENGTH, int threadsX, int threadsY)
{
	dim3 threads(threadsX, threadsY);
    dim3 blocks(iDivUp(imageW, threadsX), iDivUp(imageH, threadsY));

    hipBindTextureToArray(texSrc, src);
    convolutionRowsKernel<<<blocks, threads>>>(
        dst,
        imageW,
        imageH,
        KERNEL_RADIUS,
        KERNEL_LENGTH
    );
    hipUnbindTexture(texSrc);
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////

/////////////////////////////////////// Column convolution filter ////////////////////////////////////////////////
__global__ void convolutionColumnsKernel( unsigned char *dst, int imageW, int imageH, int KERNEL_RADIUS, int KERNEL_LENGTH)
{
    const   int ix = MAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = MAD(blockDim.y, blockIdx.y, threadIdx.y);
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

    if(ix >= imageW || iy >= imageH)
        return;

    float sum = 0;

   /* #if(UNROLL_INNER)
        sum = convolutionColumn<2 * KERNEL_RADIUS>(x, y);
    #else*/
    #pragma unroll
        for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
            sum += tex2D(texSrc, x, y + (float)k) * Kernel[KERNEL_RADIUS - k];
        
   // #endif

     dst[MAD(iy, imageW, ix)] = (unsigned char)sum;
}

void convolutionColumnsGPU( unsigned char *dst, hipArray *src, int imageW, int imageH, int KERNEL_RADIUS, int KERNEL_LENGTH, int threadsX, int threadsY)
{
    dim3 threads( threadsX, threadsY);
    dim3 blocks(iDivUp(imageW, threadsX), iDivUp(imageH, threadsY));

    hipBindTextureToArray(texSrc, src);
    convolutionColumnsKernel<<<blocks, threads>>>(
        dst,
        imageW,
        imageH,
        KERNEL_RADIUS,
        KERNEL_LENGTH
    );
    hipUnbindTexture(texSrc);
}

/////////////////////////////////////// Combining the two blurs ////////////////////////////////////////////////
gpu_error_t gpu_blur( gpu_context_t *ctx , int KERNEL_RADIUS)
{	
	assert(KERNEL_RADIUS);
	gpu_error_t error = GPU_OK;

	//float elapsedtime;
	//hipEvent_t start, stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	//hipEventRecord(start,0);

	int KERNEL_LENGTH = (2 * KERNEL_RADIUS + 1);
	const int imageW = ctx->width;
    const int imageH = ctx->height;
    	
	float *tempKernel;
	unsigned char *in;
	tempKernel = (float *)malloc(KERNEL_LENGTH * sizeof(float));

	in = ctx->output_buffer_1;
	
	hipArray *src;
    hipChannelFormatDesc floatTex = hipCreateChannelDesc<unsigned char>();
    hipMallocArray(&src, &floatTex, imageW, imageH);
    
    unsigned char *tempOutput;
    hipMalloc((void **)&tempOutput, imageW * imageH );   
    
	error = checkCudaError();
	
	////////////// calculating kernel //////////////
	float sum = 0;
	float sig = 1/3;
	float dist;
    for(int i = 0; i < KERNEL_LENGTH; i++)
    {
		//dist = (float)i - KERNEL_RADIUS;
    	dist = (float)(i - KERNEL_RADIUS) / (float)KERNEL_RADIUS;
    	tempKernel[i] = expf(- dist * dist / 2); //exponential decay, e^(-dist)
		//tempKernel[i] = (1/(sig*sqrtf(2*M_PI)))*(expf(-.5*pow(dist/sig, 2))); //guassian kernel
    	sum += tempKernel[i];
    }
    for(int i = 0; i < KERNEL_LENGTH; i++)
        tempKernel[i] /= tempKernel[0];            
	hipMemcpyToSymbol(HIP_SYMBOL(Kernel), tempKernel, KERNEL_LENGTH * sizeof(float));       
	////////////////////////////////////////////////

    hipMemcpyToArray(src, 0, 0, in, imageW * imageH, hipMemcpyHostToDevice);
    convolutionRowsGPU( tempOutput, src, imageW, imageH, KERNEL_RADIUS, KERNEL_LENGTH, ctx->threadsX, ctx->threadsY);


 	if(checkCudaError() == GPU_OK)   
 	{
    	hipMemcpyToArray(src, 0, 0, tempOutput, imageW * imageH, hipMemcpyDeviceToDevice);
    	convolutionColumnsGPU( tempOutput, src, imageW, imageH, KERNEL_RADIUS, KERNEL_LENGTH, ctx->threadsX, ctx->threadsY);

	}

	hipMemcpy(in, tempOutput, imageW * imageH, hipMemcpyDeviceToHost);
	hipMemcpy( ctx->gpu_buffer_1, tempOutput, imageW * imageH, hipMemcpyDeviceToDevice);	// This is needed so that next filter is able to use gpu_buffer_1
	error = checkCudaError();

	hipFree(tempOutput);
	hipFreeArray(src);

	//hipEventRecord(stop,0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&elapsedtime,start,stop);
	//hipEventDestroy(start);
	//hipEventDestroy(stop);
	
	//FILE *file;
	//file = fopen("../timing.txt","a+");
	//fprintf(stderr,"Smoothing:%lf \n",elapsedtime);
	//fclose(file);
	
	return error;
	
}
