#include "hip/hip_runtime.h"
/*
// This source file contains the Cuda Code for grayscale of a source Image.
// It is a part of Cuda Image Processing Library .
// Copyright (C) 2011 Remaldeep Singh

// This library is free software; you can redistribute it and/or
// modify it under the terms of the GNU Lesser General Public
// License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.

// This library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// Lesser General Public License for more details.

// You should have received a copy of the GNU Lesser General Public
// License along with this library; if not, write to the Free Software
// Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA
*/

#include "assert.h"
#include "stdio.h"
#include "hip/hip_runtime.h"
#include "../API/gpu_filter_api.h"

/////////////// Grayscale Cuda Fucntion ////////////////////
__global__ void grayscaleKernel(int width, int height, unsigned char *gpu_in_1, unsigned char *gpu_in_4)
{
	int tx = threadIdx.x + (blockIdx.x * blockDim.x);
	int ty = threadIdx.y + (blockIdx.y * blockDim.y);
	int offset = tx + ty * blockDim.x * gridDim.x;
	//int th_value = 40;

	if(offset < width * height)
	{
		float color = 0.3 * (gpu_in_4[offset * 4 + 0]) + 0.6 * (gpu_in_4[offset * 4 + 1]) + 0.1 * (gpu_in_4[offset * 4 + 2]);
		gpu_in_1[offset] = (unsigned char)color;
	}

}

///////////////// CUDA function call wrapper /////////////////
gpu_error_t gpu_grayscale(gpu_context_t *ctx)
{
	assert(ctx);
	
	float elapsedtime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	gpu_error_t error = GPU_OK;

	int threadsX = ctx->threadsX;
	int threadsY = ctx->threadsY;
	int temp1 = ((ctx->width % threadsX) != 0 ? (ctx->width / threadsX) + 1 : ctx->width / threadsX );
	int temp2 = ((ctx->height % threadsY) != 0 ? (ctx->height / threadsY) + 1 : ctx->height / threadsY );

	////////////////////////// Kernel Call //////////////////////////////////	
	dim3 block(threadsX,threadsY);
	dim3 grid(temp1,temp2);
	grayscaleKernel<<<grid,block>>>( ctx->width, ctx->height, ctx->gpu_buffer_1, ctx->gpu_buffer_4);
	/////////////////////////////////////////////////////////////////////////////////

	if( hipSuccess != hipMemcpy(ctx->output_buffer_1, ctx->gpu_buffer_1, ctx->width * ctx->height , hipMemcpyDeviceToHost))
		error = GPU_ERR_MEM;

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//FILE *file;
	//file = fopen("../timing.txt","a+");
	//fprintf(stderr,"\nGrayscale:%lf \n",elapsedtime);
	//fclose(file);
	
	return error;
}

