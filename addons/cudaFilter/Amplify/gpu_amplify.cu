#include "hip/hip_runtime.h"
/*
// This source file contains the Cuda Code for Amplification of a source Image.
// It is a part of Cuda Image Processing Library .
// Copyright (C) 2011 Remaldeep Singh

// This library is free software; you can redistribute it and/or
// modify it under the terms of the GNU Lesser General Public
// License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.

// This library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// Lesser General Public License for more details.

// You should have received a copy of the GNU Lesser General Public
// License along with this library; if not, write to the Free Software
// Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA
*/

#include "hip/hip_runtime.h"
#include "../API/api.h"
#include "assert.h"
#include "stdio.h"

__global__ void amplify( unsigned char *in, float ampValue, int size)
{
	int 	 x = threadIdx.x + __mul24(blockIdx.x,blockDim.x);
	int      y = threadIdx.y + __mul24(blockIdx.y,blockDim.y);
	int offset = x + y * __mul24(blockDim.x, gridDim.x);
	
	float temp;

	if( offset < size )
	{
		temp = (float)in[offset];
		in[offset] = ((temp * (float)ampValue) > 255) ? 255 : (unsigned char)(temp * (float)ampValue);
	}
}


gpu_error_t gpu_amplify(gpu_context_t *ctx, float ampValue)
{
	assert(ampValue);
	assert(ctx);

	float elapsedtime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	gpu_error_t error = GPU_OK;
	int threadsX = ctx->threadsX;
	int threadsY = ctx->threadsY;
	int temp1 = ((ctx->width % threadsX) != 0 ? (ctx->width / threadsX) + 1 : ctx->width / threadsX );
	int temp2 = ((ctx->height % threadsY) != 0 ? (ctx->height / threadsY) + 1 : ctx->height / threadsY );

	/////////////////////////////// Amplify call ////////////////////////////////////
	dim3 threads(threadsX,threadsY);
	dim3 blocks(temp1,temp2);	
	amplify<<<blocks,threads>>>( ctx->gpu_buffer_1, ampValue, (ctx->width * ctx->height));	
	//////////////////////////////////////////////////////////////////////////////////

	if(hipSuccess != hipMemcpy( ctx->output_buffer_1, ctx->gpu_buffer_1, ctx->width * ctx->height, hipMemcpyDeviceToHost))
		error = GPU_ERR_MEM;

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//FILE *file;
	//file = fopen("../timing.txt","a+");
	//fprintf(stderr,"Amplify:%lf \n",elapsedtime);
	//fclose(file);
	
	return error;
}
