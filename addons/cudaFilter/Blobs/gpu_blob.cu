#include "hip/hip_runtime.h"
/*
// This source file contains the Cuda Code for Blob Detection of a source Image.
// It is a part of Cuda Image Processing Library .
// Copyright (C) 2011 Remaldeep Singh

// This library is free software; you can redistribute it and/or
// modify it under the terms of the GNU Lesser General Public
// License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.

// This library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// Lesser General Public License for more details.

// You should have received a copy of the GNU Lesser General Public
// License along with this library; if not, write to the Free Software
// Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA
*/



/*	
	In the shared memory the labelSharedMemory contains the index value of the pixel as root. Labelling is done by storing value of indices at the pixels.
	The index value stored is the one that is minimum from its neighbouring 8 pixels.
							
	-------------------------------------------------------------------------
	|							Shared Memory								|
	|	-------------------------		-----------------------------		|
	|	|						|		|							|		|
	|	|						|		|							|		|
	|	|		Label			|		|		  Segment 			|		|
	|	|	Shared Memory		|		|	   Shared Memory    	|		|
	|	|						|		|							|		|
	|	|						|		|							|		|
	|	|						|		|							|		|
	|	-------------------------		-----------------------------		|
	-------------------------------------------------------------------------

	The best way to do labelling is using disjoint set datasctructure(Union Find DS).
	See Wikipidea
	Dont Use __mul24 for devices with compute capability >= 2.0.
	Hence define the apropriate flags accordingly.
	The gpu_labels_uchar is only for debugging. Remove it when you are done.
*/


#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"
#include "../API/api.h"

inline __device__ int findRoot(int* buf, int x) 
{
	int nextX;
    do {
	  nextX = x;
      x = buf[nextX];
    } while (x < nextX);
    return x;    
}

inline __device__ void unionTrees(int* buf, unsigned char *buf_uchar, unsigned char seg1, unsigned char seg2, int reg1, int reg2, int* changed)
{
	if(seg1 == seg2) 
	{			
		int newReg1 = findRoot(buf, reg1);		
		int newReg2 = findRoot(buf, reg2);	
	
		if(newReg1 > newReg2) {			
			atomicMin(buf+newReg1, newReg2);		
			//gpuMin(buf+newReg1,newReg2);
			buf_uchar[newReg1] = min( buf_uchar[newReg1], newReg2);		
			changed[0] = 1;			
		} else if(newReg2 > newReg1) {		
			atomicMin(buf+newReg2, newReg1);	
			//gpuMin1(buf+newReg2,newReg2);
			buf_uchar[newReg2] = min( buf_uchar[newReg2], newReg2);
			changed[0] = 1;
		}			
	} 	
}

texture<unsigned char, 2, hipReadModeElementType> texSrc;


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/*													  Local labelling of Blobs 															   */
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void localLabelling( unsigned char *gpu_in, int *gpu_labels, unsigned char *gpu_labels_uchar, const int pitch, const int segOff, const int dataWidth)
{
	int 	  x = blockIdx.x * blockDim.x + threadIdx.x;
    int       y = blockIdx.y * blockDim.y + threadIdx.y;
    int  offset = x + y * blockDim.x * gridDim.x;
    int shPitch = blockDim.x + 2;	// This is the pitch for shared memory of labels.

    int    localIndex = threadIdx.x + 1 + (threadIdx.y + 1) * shPitch; // This is the local address inside shared memory that has 1 pixel width of apron.
    int      newLabel = localIndex;
    int      oldLabel = 0;
    int         index = x + y * pitch; // This is the address of the pixel in global memory
    int segLocalIndex = localIndex + segOff;

	// This is the new syntax for declaring shared memory //
    extern __shared__ int sMem[];

    //shared flag that is used to check for the final solution on the processed tile 
	//if there are any two connected elements with different labels the flag is set to 1
	__shared__ int sChanged[1];

	////// Initializing the shared memory. Setting the boundary values to 0 i.e background /////
	if(threadIdx.x == blockDim.x-1) 
	{	
		sMem[localIndex+1] = 0;
		sMem[segLocalIndex+1] = 0;
	}
	if(threadIdx.x == 0) 
	{	
		sMem[localIndex-1] = 0;
		sMem[segLocalIndex-1] = 0;
	}
	if(threadIdx.y == blockDim.y-1) {			
		sMem[localIndex+shPitch] = 0;
		sMem[segLocalIndex+shPitch] = 0;

		if(threadIdx.x == 0) {			
			sMem[localIndex+shPitch-1] = 0;
			sMem[segLocalIndex+shPitch-1] = 0;
		}
		if(threadIdx.x == blockDim.x-1) {			
			sMem[localIndex+shPitch+1] = 0;
			sMem[segLocalIndex+shPitch+1] = 0;
		}	
	}
	if(threadIdx.y == 0) {			
		sMem[localIndex-shPitch] = 0;
		sMem[segLocalIndex-shPitch] = 0;
		if(threadIdx.x == 0) {			
			sMem[localIndex-shPitch-1] = 0;
			sMem[segLocalIndex-shPitch-1] = 0;
		}
		if(threadIdx.x == blockDim.x-1) {			
			sMem[localIndex-shPitch+1] = 0;
			sMem[segLocalIndex-shPitch+1] = 0;
		}	
	}

	/// VVV IMP: I think that the variables declared inside a __global__ function call are register variables, and not normal variables.
	/// The register variables are faster than shared memory. But dont overuse it due to threads throughput.
	unsigned char pixel;
	unsigned char nPixel[8];	// The neighbouring pixels.

	// Current pixel retrieved for processing //
	//pixel = tex2D( texSrc, x, y);  // This is with usage of textures.
	pixel = gpu_in[ offset ];	// This is with global memory call.
	
	sMem[segLocalIndex] = (int)pixel;	// This step will load the segmentation shared memory with all the required pixels
	__syncthreads();

	//store data about segments into registers so that we don't have to access shared memory
	//(the data are never modified)
	nPixel[0] = sMem[segLocalIndex-shPitch-1];
	nPixel[1] = sMem[segLocalIndex-shPitch];
	nPixel[2] = sMem[segLocalIndex-shPitch+1];
	nPixel[3] = sMem[segLocalIndex-1];
	nPixel[4] = sMem[segLocalIndex+1];
	nPixel[5] = sMem[segLocalIndex+shPitch-1];
	nPixel[6] = sMem[segLocalIndex+shPitch];
	nPixel[7] = sMem[segLocalIndex+shPitch+1];

	while(!NULL)
	{
		//in first pass the newLabel is equal to the local address of the element
		sMem[localIndex] = newLabel;

		//reset the check flag for each block
		if((threadIdx.x | threadIdx.y) == 0) sChanged[0] = 0;
		oldLabel = newLabel;
		__syncthreads();

		//if the element is not a background, compare the element's label with its neighbors
		if(pixel != 0) 
		{	
			if( pixel == nPixel[0])
				newLabel = min( newLabel, sMem[localIndex-shPitch-1]);
			if( pixel == nPixel[1])
				newLabel = min( newLabel, sMem[localIndex-shPitch]);
			if( pixel == nPixel[2])
				newLabel = min( newLabel, sMem[localIndex-shPitch+1]);
			if( pixel == nPixel[3])
				newLabel = min( newLabel, sMem[localIndex-1]);
			if( pixel == nPixel[4])
				newLabel = min( newLabel, sMem[localIndex+1]);
			if( pixel == nPixel[5])
				newLabel = min( newLabel, sMem[localIndex+shPitch-1]);
			if( pixel == nPixel[6])
				newLabel = min( newLabel, sMem[localIndex+shPitch]);
			if( pixel == nPixel[7])
				newLabel = min( newLabel, sMem[localIndex+shPitch+1]);
				
		}
		__syncthreads();

		if( oldLabel > newLabel)
		{
			//if there is a neigboring element with a smaller label, update the equivalence tree of the processed element
			//(the tree is always flattened in this stage so there is no need to use findRoot to find the root)	
			//VVVVIMP: This step is like merging of two trees together.				
            //Be carefull when removing this function. Atomic is used to prevent multiple threads from accessing same memory.
			//It is like a particualar thread has acquired a lock on the address.	
					
			atomicMin(sMem+oldLabel, newLabel); 

			//sMem[localIndex] = newLabel;
			//sMem[oldLabel] = min( sMem[oldLabel], newLabel);
			
			//set the flag to 1 bcoz it is necessary to perform another iteration of the CCL solver
			sChanged[0] = 1;
		}
		__syncthreads();

		if(sChanged[0] == 0) break;

		//flatten the equivalence tree
		newLabel = findRoot(sMem,newLabel);			
		__syncthreads();
	
	}	

	if(pixel == 0) newLabel = -1;	 // This is the labelling of the background pixel.
	else
	{	// The following loop translates each local label to a unique global label.
		//transfer the label into global coordinates 
		y = newLabel / (blockDim.x+2);
		x = newLabel - y*(blockDim.x+2);
		x = blockIdx.x*blockDim.x + x-1;
		y = blockIdx.y*blockDim.y + y-1;
		newLabel = x+y*dataWidth;	
	}	

	gpu_labels[index] = newLabel;
	gpu_labels_uchar[index] = (unsigned char)newLabel;

}

/////////////////////////////////////////////// Merge Borders ////////////////////////////////////////////////////////////////////////

__global__ void mergeEquivalenceTreesOnBordersKernel( int* dLabelsInOut, unsigned char *gpu_labels_uchar, const int pitch, const int tileDim);

void mergeBorders( int *gpu_labels, unsigned char *gpu_labels_uchar, int threadsX, int threadsY, int imageW, int imageH)
{

/*
	int xTiles = 4;
	int yTiles = 4;
	int threadsPerBlock = threadsX;	// This denotes the no. of pixels in borders to be merged at a time.... If the size of border is large we can also increment these threads
	int tileSize = threadsX;

	dim3 block(xTiles,yTiles,threadsPerBlock);
	dim3 grid(imageW/(block.x*block.z), imageH/(block.y*block.z));

	//merge<<<grid,block>>>( gpu_labels, gpu_labels_uchar, tileSize, imageW);
	//mergeEquivalenceTreesOnBordersKernel<<<grid,block>>>(gpu_labels, gpu_labels_uchar, imageW, tileSize);
*/

	int tileSize = threadsX;
	size_t offset;
			
	while(tileSize < imageW || tileSize < imageH) 
	{
		//compute the number of tiles that are going to be merged in a singe thread block
		int xTiles = 4;
		int yTiles = 4;
		if(xTiles*tileSize > imageW) xTiles = imageW / tileSize;
		if(yTiles*tileSize > imageH) yTiles = imageH / tileSize;
		//the number of threads that is going to be used to merge neigboring tiles
		int threadsPerBlock = threadsX;
		if(tileSize < threadsPerBlock) threadsPerBlock = tileSize;
		dim3 block(xTiles,yTiles,threadsPerBlock);
		dim3 grid(imageW/(block.x*tileSize), imageH/(block.x*tileSize), 1);
//fprintf(stderr,"I was here\n");
		//call KERNEL 2
		mergeEquivalenceTreesOnBordersKernel<<<grid, block>>>( gpu_labels, gpu_labels_uchar, imageW, tileSize);
//fprintf(stderr,"I was here %d %d %d %dx %dy \n", tileSize, grid.x, grid.y, xTiles, yTiles);
		if(yTiles > xTiles) tileSize = yTiles * tileSize;
		else tileSize = xTiles * tileSize;
				
	}

}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void mergeEquivalenceTreesOnBordersKernel( int* dLabelsInOut, unsigned char *gpu_labels_uchar, const int pitch, const int tileDim)
{
	//local tileX and Y are stored directly in blockIdx.x and blockIdx.x
	//all threads for each block are stored in the z-dir of each block (threadIdx.z)
	int tileX = threadIdx.x + blockIdx.x * blockDim.x;	
	int tileY = threadIdx.y + blockIdx.y * blockDim.y;
	//the number of times each thread has to be used to process one border of the tile
	int threadIterations = tileDim / blockDim.z;
	//dimensions of the tile on the next level of the merging scheme
	int nextTileDim = tileDim * blockDim.x;
	
	unsigned char seg;
	int offset;
	
	//shared variable that is set to 1 if an equivalence tree was changed
	__shared__ int sChanged[1];
	while(1) {		
		//reset the check variable
		if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
			sChanged[0] = 0;			
		 }		
		__syncthreads();
		//first process horizontal borders that are between merged tiles (so exclude tiles from the last row)
		if(threadIdx.y < blockDim.y-1) {
			//the horizontal border corresponds to the last row of the tile
			unsigned int y = (tileY+1)*tileDim-1;	
			//offset of the element from the left most boundary of the tile
			offset = threadIdx.x*tileDim + threadIdx.z;
			unsigned int x = tileX*tileDim + threadIdx.z;
#pragma	unroll			
			for(int i=0;i<threadIterations;++i) {					
				//load the segment data for the element
				seg = tex2D(texSrc, x, y); 
				if(seg != 0) {		
					//address of the element in the global space
					int idx = x+y*pitch;				
					//perform the unionTrees operation on neigboring elements from other tiles that are to be merged with the processed tile
					if(offset>0) unionTrees(dLabelsInOut, gpu_labels_uchar, seg, tex2D(texSrc, x-1, y+1), idx, idx-1+pitch, sChanged);
					unionTrees(dLabelsInOut, gpu_labels_uchar, seg, tex2D(texSrc, x, y+1), idx, idx+pitch, sChanged);
					if(offset<nextTileDim-1) unionTrees(dLabelsInOut, gpu_labels_uchar, seg, tex2D(texSrc, x+1, y+1), idx, idx+1+pitch, sChanged);
					
				}
				//set the processed element to the next in line on the same boundary (in case the threads are used for multiple elements on the boundary)
				x += blockDim.z;
				offset += blockDim.z;
			}
		}
		// vertical right borders
		if(threadIdx.x < blockDim.x-1) 
		{
			unsigned int x = (tileX+1)*tileDim-1;		
			offset = threadIdx.y*tileDim + threadIdx.z;
			unsigned int y = tileY*tileDim+threadIdx.z;
#pragma unroll
			for(int i=0;i<threadIterations;++i) 
			{			
				seg = tex2D(texSrc, x, y); 
				if(seg != 0) 
				{
					int idx = x+y*pitch;
					//perform the unionTrees operation on neigboring elements from other tiles that are to be merged with the processed tile
					if(offset>0) unionTrees(dLabelsInOut, gpu_labels_uchar, seg, tex2D(texSrc, x+1, y-1), idx, idx+1-pitch, sChanged);
					unionTrees(dLabelsInOut, gpu_labels_uchar, seg, tex2D(texSrc, x+1, y), idx, idx+1, sChanged);
					if(offset<nextTileDim-1) unionTrees(dLabelsInOut, gpu_labels_uchar, seg, tex2D(texSrc, x+1, y+1), idx, idx+1+pitch, sChanged);			
				}	
				y += blockDim.z;
				offset += blockDim.z;
			}		
		}		
		__syncthreads();
		
		if(sChanged[0] == 0) 		
			break;	
		
		__syncthreads();
	}	
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/*													Flattening of all the elements															*/
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void flattenEquivalenceTreesKernel(int* gpu_labels_out, int* gpu_labels_in, unsigned char *gpu_labels_uchar, unsigned int pitch, const int dataWidth, int *gpu_nRegions)												
{
	unsigned int     x = (blockIdx.x*blockDim.x)+threadIdx.x;
    unsigned int     y = (blockIdx.y*blockDim.y)+threadIdx.y;  
    unsigned int index = x+y*pitch;
    unsigned int label = gpu_labels_in[index];

	unsigned int newLabel;

	if((label != -1) && (label != index))
	{
		newLabel = findRoot( gpu_labels_in, label);

		if(newLabel < label)
		{
			gpu_labels_out[index] = newLabel;
			gpu_labels_uchar[index] = (unsigned char)newLabel;
		}
	}
	__syncthreads();

	if(gpu_labels_out[index] == index)
	{
		atomicAdd( gpu_nRegions, 1);
	}
	
}

void flattenTrees( int *gpu_labels, unsigned char *gpu_labels_uchar, int threadsX, int threadsY, int imageW, int imageH, int *gpu_nRegions)
{	
	dim3 block(threadsX, threadsY, 1);
    dim3 grid(imageW / block.x, imageH / block.y, 1);

	hipMemset( gpu_nRegions, 0, sizeof(int));
    flattenEquivalenceTreesKernel<<<grid,block>>>( gpu_labels, gpu_labels, gpu_labels_uchar, imageW, imageW, gpu_nRegions);
    
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* 													 Calculating centroid 															  */
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// FIXME: Try and make it volatile for masively random access in case of blob centroid.

__global__ void calcCentroidKernel( int *gpu_labels, int *gpu_nRegions, int *gpu_regionOff, int *gpu_regionSize, int *gpu_centroid, int *i, int pitch)
{
	int     x = (blockIdx.x*blockDim.x)+threadIdx.x;
    int     y = (blockIdx.y*blockDim.y)+threadIdx.y;  
    int index = x+y*pitch;
    int     j = 0;

	//// Implement this critical section with atomics. The followning code wont work... :(
	if( gpu_labels[index] == index)
	{
		atomicAdd( i, 1);						
		atomicAdd( gpu_regionOff + (*i), index);		// Change i to *i
	}

	// I dont think there is a need to __syncthreads but try it, if it doesnt work out.

	// Finding the index where offset is stored.	
	for( j=0; j < *gpu_nRegions; j++)
	{
		if( gpu_labels[index] == gpu_regionOff[j])
			break;
	}
	
	// Storing the value of centroid at 2*j position.
	if( gpu_labels[index] != -1)
	{
		atomicAdd( gpu_centroid+(2*j), x);
		atomicAdd( gpu_centroid+(2*j)+1, y);
		atomicAdd( gpu_regionSize+j, 1);
	}	

}

__global__ void calcCentroidSharedKernel(int *gpu_labels,int *gpu_nRegions,int *gpu_regionOff,int *gpu_regionsSize,int *gpu_centroid,int *i,int pitch)
{

	extern __shared__ int sMem[];

	int     			x = (blockIdx.x*blockDim.x)+threadIdx.x;
    int     			y = (blockIdx.y*blockDim.y)+threadIdx.y;  
    int 			index = x+y*pitch;
    int 				j = 0; 
    int 	shSize_Offset = 0;
    int shCentroid_Offset = *gpu_nRegions;

	if((index >= 0) && (index < (3*(*gpu_nRegions))))
		sMem[index] = 0;
    
    if( gpu_labels[index] == index)
    {
		atomicAdd( i, 1);
		atomicAdd( gpu_regionOff + (*i), index);
    }
	__syncthreads();
	
    for( j=0; j < *gpu_nRegions; j++)
	{
			if( gpu_labels[index] == gpu_regionOff[j])
			{
						atomicAdd( sMem+shCentroid_Offset+(2*j), x);
						atomicAdd( sMem+shCentroid_Offset+(2*j)+1, y);
						atomicAdd( sMem+j, 1);
			}
	}
	__syncthreads();

	if((threadIdx.x == 0) && (threadIdx.y == 0))
	{
		atomicAdd(gpu_centroid+(2*j), sMem[shCentroid_Offset+(2*j)]);
		atomicAdd(gpu_centroid+(2*j)+1, sMem[shCentroid_Offset+(2*j)+1]);
		atomicAdd(gpu_regionsSize+j, sMem[j]);
	}

}

		// nRegions was added later delete it..............

void calcCentroid( int *gpu_labels, unsigned char *gpu_labels_uchar, int threadsX, int threadsY, int imageW, int imageH, int nRegions, int *gpu_nRegions, int *gpu_regionOff, int *gpu_regionSize, int *gpu_centroid)
{
	int *i;
	hipMalloc( (void **)&i, sizeof(int));
	hipMemset( i, -1, sizeof(int));
	int shSize = (nRegions)*4*sizeof(int);
	dim3 block( 15, 16, 1);
	dim3 grid( imageW / block.x, imageH / block.y, 1);

	calcCentroidKernel<<<grid,block>>>( gpu_labels, gpu_nRegions, gpu_regionOff, gpu_regionSize, gpu_centroid, i, imageW);
	

/*	if(shSize != 0)
	{
		calcCentroidSharedKernel<<<grid,block,shSize>>>(gpu_labels,gpu_nRegions,gpu_regionOff,gpu_regionSize,gpu_centroid,i,imageW);
	}
*/
}	

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/*                 									Main Wrapper about the function   								  				  */
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
gpu_error_t gpu_DetectBlob( gpu_context_t *ctx)
{

	gpu_error_t err = GPU_OK;

	int *regionSize, *centroid;
//	int min_blobSize = 0;
//	int max_blobSize = 1000;

	int   imageW = ctx->width;
	int   imageH = ctx->height;
	int threadsX = ctx->threadsX;
	int threadsY = ctx->threadsY;

	fprintf(stderr,"%d %d dimensions \n",ctx->width,ctx->height);

	float elapsedtime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int *gpu_nRegions, *nRegions;
	hipMalloc( (void **)&gpu_nRegions, sizeof(int));
	nRegions = (int *)malloc(sizeof(int));
	
    int *gpu_labels;
    hipMalloc( (void **)&gpu_labels, imageW * imageH * sizeof(int));

	err = checkCudaError();
	if( err != GPU_OK)
		return err;

    int *labels_int;
    labels_int = (int *)malloc(imageW*imageH*sizeof(int));

    //// This buffer is temporary and only used for debuggin purpose and is reponsible for diplaying the last detected blob image ////
    unsigned char *gpu_labels_uchar;
    hipMalloc( (void **)&gpu_labels_uchar, imageW * imageH * sizeof(unsigned char));

   	err = checkCudaError();
	if( err != GPU_OK)
		return err;
	//////
   
    hipEventRecord(start,0);

    /******************************************* Local Shared Labelling ****************************************************/
    dim3 threads(threadsX,threadsY);
    dim3 blocks( imageW/threadsX, imageH/threadsY);

    int labelSize = (threads.x + 2) * (threads.y + 2) * sizeof(int); //This is the size for storage of labels to the corresponding pixels
    int   segSize = (threads.x + 2) * (threads.y + 2) * sizeof(int); //This is the size of storage for segments.
    
	localLabelling<<< blocks, threads, (labelSize + segSize)>>>( ctx->gpu_buffer_1, gpu_labels, gpu_labels_uchar, imageW, labelSize/sizeof(int), imageW);

	err = checkCudaError();
	if( err != GPU_OK)
		return err;
	/***********************************************************************************************************************/

	
	/******************************************* Merging Blobs Together ****************************************************/

	hipArray *src;
    hipChannelFormatDesc floatTex = hipCreateChannelDesc<unsigned char>();
    hipMallocArray(&src, &floatTex, imageW, imageH);
    hipMemcpyToArray(src, 0, 0, ctx->gpu_buffer_1, imageW * imageH, hipMemcpyDeviceToDevice);
    hipBindTextureToArray(texSrc, src);  

	mergeBorders( gpu_labels, gpu_labels_uchar, threadsX, threadsY, imageW, imageH);

	err = checkCudaError();
	if( err != GPU_OK)
		return err;
 
    hipUnbindTexture(texSrc);


	/****************************************** Updating all the labels (i.e flattening) ********************************************/

	flattenTrees( gpu_labels, gpu_labels_uchar, threadsX, threadsY, imageW, imageH, gpu_nRegions);
	hipMemcpy( nRegions, gpu_nRegions, sizeof(int), hipMemcpyDeviceToHost);
	err = checkCudaError();
	if( err != GPU_OK)
		return err;


	int *gpu_regionOff, *gpu_regionSize, *gpu_centroid;
	hipMalloc( (void **)&gpu_regionOff, (*nRegions)*sizeof(int));
	hipMalloc( (void **)&gpu_regionSize, (*nRegions)*sizeof(int));
	hipMalloc( (void **)&gpu_centroid, (*nRegions)*2*sizeof(int));
	hipMemset( gpu_regionOff, 0, (*nRegions)*sizeof(int));
	hipMemset( gpu_regionSize, 0, (*nRegions)*sizeof(int));
	hipMemset( gpu_centroid, -1, (*nRegions)*2*sizeof(int));
	regionSize = (int *)malloc((*nRegions)*sizeof(int));
    centroid = (int *)malloc((*nRegions)*2*sizeof(int));

	/********************************************************** Calculating Centroid *******************************************************/

	calcCentroid( gpu_labels, gpu_labels_uchar, threadsX, threadsY, imageW, imageH, *nRegions, gpu_nRegions, gpu_regionOff, gpu_regionSize, gpu_centroid);
	err = checkCudaError();
	if( err != GPU_OK)
		return err;


	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	fprintf(stderr,"Blobs: %lf\n\n",elapsedtime);

	
	hipMemcpy( regionSize, gpu_regionSize, (*nRegions)*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy( centroid, gpu_centroid, (*nRegions)*2*sizeof(int), hipMemcpyDeviceToHost);	
	// In Order to find a centroid just divide centroid[i] and centroid[i+1] with regionSize[i] to get X and Y respectively. 
	
	hipMemcpy( ctx->output_buffer_1, gpu_labels_uchar, imageW*imageH, hipMemcpyDeviceToHost);
	hipMemcpy( labels_int, gpu_labels, imageW*imageH*sizeof(int), hipMemcpyDeviceToHost);
	err = checkCudaError();
	if( err != GPU_OK)
		return err;
    hipMemcpy( labels_int, gpu_labels, imageW*imageH*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(gpu_labels);
	hipFree(gpu_labels_uchar);

	// Rest of the code is just for debugging. Remove it after measuring performance. //
/*
	FILE *file;
	file = fopen("debug_1.txt","a+"); // apend file (add text to a file or create a file if it does not exist.
	for(int i=0;i<imageW*imageH;i++)
	{
		if((i>imageW-1) && (i%imageW==0))
			fprintf(file,"\n");
		fprintf(file,"%d ", labels_int[i]); 
	}
	fprintf(file,"\n");
	fclose(file); //done!
*/

	FILE *file;
	file = fopen("regions.txt","a+"); // apend file (add text to a file or create a file if it does not exist.
	fprintf(file,"%d %f ", *nRegions, elapsedtime);
	
	for(int i=0;i<(*nRegions);i++)
	{
		fprintf(file," %d %d ", centroid[2*i],centroid[2*i]+1);
		fprintf(file,"%d ", regionSize[i]);
	}

	fprintf(file,"\n");
	fclose(file); //done!

	return err;
}
