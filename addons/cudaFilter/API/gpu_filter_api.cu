/*
// This source file contains the API code various gpu methods provided by the library.
// It is a part of Cuda Image Processing Library ).
// Copyright (C) 2011 Remaldeep Singh

// This library is free software; you can redistribute it and/or
// modify it under the terms of the GNU Lesser General Public
// License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.

// This library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// Lesser General Public License for more details.

// You should have received a copy of the GNU Lesser General Public
// License along with this library; if not, write to the Free Software
// Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA
*/

#include "assert.h"

#ifndef _CUDA_H
#define _CUDA_H
#include "hip/hip_runtime.h"
#endif

#include <stdio.h>
#include <assert.h>
#include <string.h>

#include "gpu_filter_api.h"

gpu_error_t last_error = GPU_OK;
hipError_t last_cuda_error = hipSuccess;

///////////////////////////////// Necessary Cuda calls /////////////////////////////////////////

/////////////// This call copies data from global memory ////////////////
void cuda_set_input(gpu_context_t *ctx, unsigned char *idata)
{
	int i = 0,
	size = ctx->width * ctx->height;

	switch ( ctx->nchannels )
	{
		case 1:
			for( ; i < size; i++)
			{
				ctx->output_buffer_1[i] = idata[i];
				// FIXME if the image is already gray (1 channel),
				// do we need to do that ? @ mathieu, No we dont need the rest of them.....
				//ctx->output_buffer_4[i * 4 + 1] = idata[i];
				//ctx->output_buffer_4[i * 4 + 2] = idata[i];
			}
			hipMemcpy( ctx->gpu_buffer_1, ctx->output_buffer_1, size, hipMemcpyHostToDevice);
			break;
		case 3:
			for( ; i < size; i++)
			{
				ctx->output_buffer_4[i * 4 + 0] = idata[i * 3 + 0];
				ctx->output_buffer_4[i * 4 + 1] = idata[i * 3 + 1];
				ctx->output_buffer_4[i * 4 + 2] = idata[i * 3 + 2];
			}
			hipMemcpy(ctx->gpu_buffer_4, ctx->output_buffer_4, size * 4, hipMemcpyHostToDevice);
			break;
		case 4:
			for( ; i < size; i++)
			{
				ctx->output_buffer_4[i * 4 + 0] = idata[i * 4 + 0];
				ctx->output_buffer_4[i * 4 + 1] = idata[i * 4 + 1];
				ctx->output_buffer_4[i * 4 + 2] = idata[i * 4 + 2];
				ctx->output_buffer_4[i * 4 + 3] = idata[i * 4 + 3];
			}
			hipMemcpy(ctx->gpu_buffer_4, ctx->output_buffer_4, size * 4, hipMemcpyHostToDevice);
			break;

		default:
			// this is because we don't know how to copy this input image to gpu buffer
			assert(0);
	}

	checkCudaError();
}
/////////////////////////////////////////////////////////////////////////

////// This code will return error occured on GPU in a string format ////
const char *gpu_error()
{
	// reset the error for next call
	gpu_error_t error = last_error;
	hipError_t cuda_error = last_cuda_error;
	last_error = GPU_OK;
	last_cuda_error = hipSuccess;

	switch (error)
	{
		case GPU_OK:
			return "OK";
		case GPU_ERR_MEM:
			return "Memory allocation";
		case GPU_ERR_CUDA:
			return hipGetErrorString(cuda_error);
	}

	return "Unknown";
}

//////////// This code will check for any Cuda related error ////////////
gpu_error_t checkCudaError()
{
	last_cuda_error = hipGetLastError();
	if (last_cuda_error != hipSuccess)
		return GPU_ERR_CUDA;
	return GPU_OK;
}

//////////// This code will create a gpu context /////////////
gpu_error_t gpu_context_create(gpu_context_t **ctx)
{
	last_error = GPU_OK;
	assert(ctx != NULL);

	// create the context and initialize it
	*ctx = (gpu_context_t *)malloc( sizeof(gpu_context_t) );
	if (*ctx == NULL)
		last_error = GPU_ERR_MEM;
	else
		memset(*ctx, 0, sizeof(gpu_context_t));

	return last_error;

}

/////////////////////////////// This code will initialize the previously created contex ///////////////////////////////////////////////////
gpu_error_t gpu_context_init(gpu_context_t *ctx, int host_height, int host_width, int host_nchannels, gpu_context_memory_t host_flag)
{
	assert(ctx != NULL);
	assert(host_height > 0);
	assert(host_width > 0);
	assert(host_nchannels == 3 || host_nchannels == 1);

	ctx->height = host_height;
	ctx->width = host_width;
	ctx->nchannels = host_nchannels;
	ctx->mem_flag = host_flag;
	// whatever the source channels is, we always use 4 channels images
	ctx->size = ctx->height * ctx->width * 4 * sizeof(unsigned char);

	hipMalloc( (void **)&ctx->gpu_buffer_4, ctx->size);
	last_error = checkCudaError();
	if(last_error == GPU_OK)
	{
		hipMalloc( (void **)&ctx->gpu_buffer_1, (ctx->height * ctx->width * sizeof(unsigned char)) );
		last_error = checkCudaError();
	}
	if(last_error == GPU_OK)
	{
		switch (ctx->mem_flag)
		{
			case GPU_MEMORY_HOST:
				ctx->output_buffer_4 = (unsigned char *)malloc(ctx->size);
				if(!(ctx->output_buffer_4))
					last_error = GPU_ERR_MEM;
				ctx->output_buffer_1 = (unsigned char *)malloc((ctx->width * ctx->height * sizeof(unsigned char)));
				if(!(ctx->output_buffer_1))
					last_error = GPU_ERR_MEM;
				break;

			case GPU_MEMORY_PINNED_WRITE_COMBINED:
				hipHostAlloc( (void **)&ctx->output_buffer_4, ctx->size, hipHostMallocWriteCombined);
				last_error = checkCudaError();
	
				if(last_error == GPU_OK)
				{
					hipHostAlloc( (void **)&ctx->output_buffer_1, (ctx->width * ctx->height * sizeof(unsigned char)), hipHostMallocWriteCombined);
					last_error = checkCudaError();
				}
				break;

			case GPU_MEMORY_PINNED:
				hipHostAlloc((void **)&ctx->output_buffer_4, ctx->size, hipHostMallocDefault);
				last_error = checkCudaError();

				if(last_error == GPU_OK)
				{
					hipHostAlloc( (void **)&ctx->output_buffer_1, (ctx->width * ctx->height * sizeof(unsigned char)), hipHostMallocDefault);
					last_error = checkCudaError();
				}
				break;

			default:
				// should never happen
				assert(0);
				last_error = GPU_ERR_MEM;
				break;
		}
	}

	/*		Calculating appropriate no. of threaqds for current dimension		*/
	int   imageW = ctx->width;
	int   imageH = ctx->height;
	int    temp1 = imageW/4;
	int    temp2 = imageH/4;	
	int 	   i = 15;

	if( (imageW==480 || imageW==240 || imageW==320 ||  imageW==640 ) && (imageH==320 || imageH==640 || imageH==240 || imageH==480))
	{
		ctx->threadsX = 20;
		ctx->threadsY = 20;
	}
	else if( (imageW==768 || imageW == 640) && (imageH==1024 || imageH == 512))
	{
		ctx->threadsX = 16;
		ctx->threadsY = 16;
	}
	else if(imageW==1280 && imageH==1024)
	{
		ctx->threadsX = 32;
		ctx->threadsY = 32;
	}
	else
	{
		/*
		while( (temp1%i != 0) && (temp2%i != 0))
		{
			i++;
			if(i>20)break;
		}
		if( i>20 )
		{
			fprintf(stderr,"Invalid dimensions for blob detection");
			exit(EXIT_FAILURE);
		}
		ctx->threadsX = i;
		ctx->threadsY = i;			
		*/
		fprintf(stderr,"Invalid dimension");	
	}

	fprintf(stderr,"%d %d threads \n",ctx->threadsX,ctx->threadsY);
	

	return last_error;
}

///////////// This code will set the context buffer to the input buffer //////////////
gpu_error_t gpu_set_input( gpu_context_t *ctx, unsigned char *idata)
{
	assert( ctx || idata );
	cuda_set_input(ctx, idata);
	return GPU_OK;
}

///////////// This code will set the input buffer to the context buffer //////////////
gpu_error_t gpu_get_output(gpu_context_t *ctx, unsigned char **output)
{
	assert( ctx );
	assert( output != NULL );

	//hipMemcpy(ctx->output_buffer_1, ctx->gpu_buffer_1, ctx->width * ctx->height , hipMemcpyDeviceToHost);
	last_error = checkCudaError();
	//if ( last_error == GPU_OK )
		*output = ctx->output_buffer_1;
	

	return last_error;
}

///// This code will deallocate all the memory held by context, including the memory on GPU //////
void gpu_context_free( gpu_context_t *ctx)
{
	assert(ctx);
	switch ( ctx->mem_flag )
	{
		case GPU_MEMORY_HOST:
			free(ctx->output_buffer_4);
			free(ctx->output_buffer_1);
			hipHostFree(ctx->gpu_buffer_4);
			hipHostFree(ctx->gpu_buffer_1);
			break;

		case GPU_MEMORY_PINNED_WRITE_COMBINED:
		case GPU_MEMORY_PINNED:
			hipHostFree(ctx->output_buffer_4);
			hipHostFree(ctx->gpu_buffer_4);
			hipHostFree(ctx->output_buffer_1);
			hipHostFree(ctx->gpu_buffer_1);
			break;

		default:
			// should never happen
			assert(0);
			break;
	}
	free(ctx);
}

void cuda_device_info(void)
{
   int  ndev;
   hipGetDeviceCount(&ndev);
   hipDeviceSynchronize();
   printf("There are %d GPUs.\n",ndev);
 
   for(int i=0;i<ndev;i++) {
     hipDeviceProp_t pdev;
     hipGetDeviceProperties(&pdev,i);
     hipDeviceSynchronize();
     printf("Name         : %s\n",pdev.name);
     printf("Capability   : %d %d\n",pdev.major,pdev.minor);
     printf("Memory Global: %d Mb\n",(pdev.totalGlobalMem+1024*1024)/1024/1024);
     printf("Memory Const : %d Kb\n",pdev.totalConstMem/1024);
     printf("Memory Shared: %d Kb\n",pdev.sharedMemPerBlock/1024);
     printf("Clock        : %.3f GHz\n",pdev.clockRate/1000000.0);
     printf("Processors   : %d\n",pdev.multiProcessorCount);
     printf("Cores        : %d\n",8*pdev.multiProcessorCount);
     printf("Warp         : %d\n",pdev.warpSize);
     printf("Max Thr/Blk  : %d\n",pdev.maxThreadsPerBlock);
     printf("Max Blk Size : %d %d %d\n",pdev.maxThreadsDim[0],pdev.maxThreadsDim[1],pdev.maxThreadsDim[2]);
     printf("Max Grid Size: %d %d %d\n",pdev.maxGridSize[0],pdev.maxGridSize[1],pdev.maxGridSize[2]);
   }
}